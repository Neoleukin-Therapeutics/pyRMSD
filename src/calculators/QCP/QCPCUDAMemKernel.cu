/*
 * QCPCUDAMemKernel.cpp
 *
 *  Created on: Apr 13, 2013
 *      Author: victor
 */

#include "QCPCUDAMemKernel.h"
#include "kernel_functions_cuda.h"
#include "../RMSDCalculationData.h"

#define OFFSET_FOR(conformation_number, number_of_conformations) (((number_of_conformations-1)* conformation_number) - (conformation_number*(conformation_number-1))/2)

inline void checkCudaError(char* message, hipError_t error_code){
	if (error_code != 0){
		std::cout<<"Error in "<<message<<". Error code: "<<error_code<<". Exiting..."<<std::flush<<std::endl;
		exit(-1);
	}
}

QCPCUDAMemKernel::QCPCUDAMemKernel(
				RMSDCalculationData* data,
				int threads_per_block,
				int blocks_per_grid):QCPCUDAKernel( data,
													threads_per_block,
													blocks_per_grid) {
	allDeviceRMSDs = NULL;

}

QCPCUDAMemKernel::~QCPCUDAMemKernel(){}

void QCPCUDAMemKernel::matrixInit(RMSDCalculationData* data){
	
	QCPCUDAKernel::matrixInit(data);
						
	//Allocate space to store all rmsds
	checkCudaError("Malloc allDeviceRMSDs", 
			hipMalloc(
					(void **) &this->allDeviceRMSDs, 
					((data->numberOfConformations*(data->numberOfConformations-1)) / 2) * sizeof(floating_point_type)));
	
}

void QCPCUDAMemKernel::matrixEnd(int rmsds_len,
									std::vector<double>& rmsds){
	rmsds.clear();
	rmsds.resize(rmsds_len);
	
	#ifdef CUDA_PRECISION_SINGLE
	
		float* buffer = new float[rmsds_len];
		
		checkCudaError("allDeviceRMSDs copy to host",
			hipMemcpy(	buffer,
					this->allDeviceRMSDs,
					rmsds_len * sizeof(float),
					hipMemcpyDeviceToHost));
		
		for(int i = 0; i < rmsds_len; ++i){
			rmsds[i] = static_cast<double>( buffer[i] );
		}
		
		delete [] buffer;
		
	#else	
		checkCudaError("allDeviceRMSDs copy to host",
			hipMemcpy(	&(rmsds[0]),
					this->allDeviceRMSDs,
					rmsds_len * sizeof(double),
					hipMemcpyDeviceToHost));
	#endif
	
	checkCudaError("allDeviceRMSDs hipFree",
				hipFree(this->allDeviceRMSDs));
}


void QCPCUDAMemKernel::matrixOneVsFollowingFitEqualCalc(
									double* reference, 
									int reference_conformation_number, 
									double* rmsd,
									RMSDCalculationData* data){
	
	floating_point_type* tmpDeviceReference = &(this->deviceCoords[reference_conformation_number*
	                                                               data->fittingConformationLength]);
	
	// Do the calculations
	calcRMSDOfOneVsFollowing CUDA_KERNEL_DIM(this->blocks_per_grid, this->threads_per_block)(
			tmpDeviceReference,
			reference_conformation_number,
			this->deviceCoords,
			data->numberOfConformations,
			data->atomsPerFittingConformation,
			data->fittingConformationLength,
			&(this->allDeviceRMSDs[OFFSET_FOR(reference_conformation_number, data->numberOfConformations)]));
}

void QCPCUDAMemKernel::matrixOneVsFollowingFitDiffersCalc(
											double* fitReference, 
											double* calcReference,
											int reference_conformation_number, 
											double* rmsd,
											RMSDCalculationData* data){
		
	floating_point_type* tmpFitDeviceReference = &(this->deviceCoords[reference_conformation_number*
	                                                                  data->fittingConformationLength]);
	floating_point_type* tmpCalcDeviceReference = &(this->deviceCalcCoords[reference_conformation_number*
	                                                                       data->calculationConformationLength]);
	
	// Do the calculations
	calcRMSDOfOneVsFollowingFitDiffersCalc CUDA_KERNEL_DIM(this->blocks_per_grid, this->threads_per_block)(
			tmpFitDeviceReference,
			tmpCalcDeviceReference,
			reference_conformation_number,
			&(this->allDeviceRMSDs[OFFSET_FOR(reference_conformation_number, data->numberOfConformations)]),
			data->numberOfConformations,
			data->fittingConformationLength,
			data->atomsPerFittingConformation,
			deviceCoords,
			data->calculationConformationLength,
			data->atomsPerCalculationConformation,
			deviceCalcCoords);
}
